#include "hip/hip_runtime.h"
extern "C" {
	#include "sph/sph_blake.h"
	#include "sph/sph_bmw.h"
	#include "sph/sph_groestl.h"
	#include "sph/sph_skein.h"
	#include "sph/sph_jh.h"
	#include "sph/sph_keccak.h"
	#include "sph/sph_luffa.h"
	#include "sph/sph_cubehash.h"
	#include "sph/sph_shavite.h"
	#include "sph/sph_simd.h"
	#include "sph/sph_echo.h"
	}
	
	#include "miner.h"
	#include "cuda_helper.h"
	#include "cuda_x11k.h"
	
	#include <stdio.h>
	#include <memory.h>
	
	static uint32_t *d_hash[MAX_GPUS];

	static unsigned char *seed_index;

	static void processHash(void *oHash, const void *iHash, const int index, const size_t len)
	{
		switch (index)
		{
			case 0:
				sph_blake512_context ctx_blake;

				sph_blake512_init(&ctx_blake);
				sph_blake512(&ctx_blake, iHash, len);
				sph_blake512_close(&ctx_blake, oHash);
				break;
			case 1:
				sph_bmw512_context ctx_bmw;

				sph_bmw512_init(&ctx_bmw);
				sph_bmw512 (&ctx_bmw, iHash, len);
				sph_bmw512_close(&ctx_bmw, oHash);
				break;
			case 2:
				sph_groestl512_context ctx_groestl;

				sph_groestl512_init(&ctx_groestl);
				sph_groestl512 (&ctx_groestl, iHash, len);
				sph_groestl512_close(&ctx_groestl, oHash);
				break;
			case 3:
				sph_skein512_context ctx_skein;

				sph_skein512_init(&ctx_skein);
				sph_skein512 (&ctx_skein, iHash, len);
				sph_skein512_close (&ctx_skein, oHash);
				break;
			case 4:
				sph_jh512_context ctx_jh;

				sph_jh512_init(&ctx_jh);
				sph_jh512 (&ctx_jh, iHash, len);
				sph_jh512_close(&ctx_jh, oHash);
				break;
			case 5:
				sph_keccak512_context ctx_keccak;

				sph_keccak512_init(&ctx_keccak);
				sph_keccak512 (&ctx_keccak, iHash, len);
				sph_keccak512_close(&ctx_keccak, oHash);
				break;
			case 6:
				sph_luffa512_context ctx_luffa1;

				sph_luffa512_init (&ctx_luffa1);
				sph_luffa512 (&ctx_luffa1, iHash, len);
				sph_luffa512_close (&ctx_luffa1, oHash);
				break;
			case 7:
				sph_cubehash512_context ctx_cubehash1;

				sph_cubehash512_init (&ctx_cubehash1);
				sph_cubehash512 (&ctx_cubehash1, iHash, len);
				sph_cubehash512_close(&ctx_cubehash1, oHash);
				break;
			case 8:
				sph_shavite512_context ctx_shavite1;

				sph_shavite512_init (&ctx_shavite1);
				sph_shavite512 (&ctx_shavite1, iHash, len);
				sph_shavite512_close(&ctx_shavite1, oHash);
				break;
			case 9:
				sph_simd512_context ctx_simd1;

				sph_simd512_init (&ctx_simd1);
				sph_simd512 (&ctx_simd1, iHash, len);
				sph_simd512_close(&ctx_simd1, oHash);
				break;
			case 10:
				sph_echo512_context ctx_echo1;

				sph_echo512_init (&ctx_echo1);
				sph_echo512 (&ctx_echo1, iHash, len);
				sph_echo512_close(&ctx_echo1, oHash);
				break;
		}
	}

	// X11K CPU Hash
	const int HASHX11K_NUMBER_ITERATIONS = 64;
	const int HASHX11K_NUMBER_ALGOS = 11;

	extern "C" void x11khash(void *output, const void *input)
	{
		static uint32_t _ALIGN(64) hashA[64/4], hashB[64/4];
		seed_index = (unsigned char *) calloc(64, sizeof(unsigned char));

		// Iteration 0
		processHash(hashA, input, 0, 80);

		for(int i = 1; i < HASHX11K_NUMBER_ITERATIONS; i++) {
			seed_index = (unsigned char *) hashA;

			processHash(hashB, hashA, seed_index[i] % HASHX11K_NUMBER_ALGOS, 64);

			memcpy(hashA, hashB, 64);
		}

		memcpy(output, hashA, 32);
	}

	//#define _DEBUG
	#define _DEBUG_PREFIX "x11k"
	#include "cuda_debug.cuh"
	
	static bool init[MAX_GPUS] = { 0 };
	static bool use_compat_kernels[MAX_GPUS] = { 0 };

	extern "C" int scanhash_x11k(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
	{
		uint32_t *pdata = work->data;
		uint32_t *ptarget = work->target;
		const uint32_t first_nonce = pdata[19];
		const int dev_id = device_map[thr_id];
		int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 20 : 19;
		if (strstr(device_name[dev_id], "GTX 1080")) intensity = 20;
			uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity); // 19=256*256*8;
		//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		if (opt_benchmark) {
			((uint32_t*)ptarget)[7] = 0x003f;
		}
		
		if (!init[thr_id])
		{
			hipSetDevice(device_map[thr_id]);
			if (opt_cudaschedule == -1 && gpu_threads == 1) {
				hipDeviceReset();
				// reduce cpu usage
				hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
				CUDA_LOG_ERROR();
			}
			gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
	
			cuda_get_arch(thr_id);
			use_compat_kernels[thr_id] = (cuda_arch[dev_id] < 500);
			if (use_compat_kernels[thr_id])
				x11_echo512_cpu_init(thr_id, throughput);
	
			quark_blake512_cpu_init(thr_id, throughput);
			quark_bmw512_cpu_init(thr_id, throughput);
			quark_groestl512_cpu_init(thr_id, throughput);
			quark_skein512_cpu_init(thr_id, throughput);
			quark_jh512_cpu_init(thr_id, throughput);
			quark_keccak512_cpu_init(thr_id, throughput);
			qubit_luffa512_cpu_init(thr_id, throughput);
			x11_luffa512_cpu_init(thr_id, throughput); // 64
			x11_shavite512_cpu_init(thr_id, throughput);
			x11_simd512_cpu_init(thr_id, throughput); // 64
			x16_echo512_cuda_init(thr_id, throughput);
		
			CUDA_CALL_OR_RET_X(hipMallocManaged((void **) &d_hash[thr_id], (size_t) 64 * throughput), 0);
			CUDA_CALL_OR_RET_X(hipMallocManaged((void **) &seed_index, (size_t) 64 * throughput), 0);

			cuda_check_cpu_init(thr_id, throughput);
	
			init[thr_id] = true;
		}
	
		uint32_t endiandata[20];
		for (int k=0; k < 20; k++)
			be32enc(&endiandata[k], pdata[k]);
	
		quark_blake512_cpu_setBlock_80(thr_id, endiandata);
		cuda_check_cpu_setTarget(ptarget);
	
		do {
			int order = 0;

			// Iteration 0
			quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
			TRACE("blake80:");
		
			for (int i = 1; i < HASHX11K_NUMBER_ITERATIONS; i++)
			{
				hipDeviceSynchronize();
				seed_index = (unsigned char *) d_hash[thr_id];

				switch (seed_index[i] % HASHX11K_NUMBER_ALGOS)
				{
					case 0:
						quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						TRACE("blake  :");
						break;
					case 1:
						quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						TRACE("bmw    :");
						break;
					case 2:
						quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						TRACE("groestl:");
						break;
					case 3:
						quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						TRACE("skein  :");
						break;
					case 4:
						quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						TRACE("jh512  :");
						break;
					case 5:
						quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						TRACE("keccak :");
						break;
					case 6:
						x11_luffa512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						TRACE("luffa  :");
						break;
					case 7:
						x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						TRACE("cube   :");
						break;
					case 8:
						x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						TRACE("shavite:");
						break;
					case 9:
						x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						TRACE("simd   :");
						break;
					case 10:
						if (use_compat_kernels[thr_id])
							x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
						else
							x16_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); order++;

						TRACE("echo   :");
						break;
				}
			}

			*hashes_done = pdata[19] - first_nonce + throughput;
	
			work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);

			if (work->nonces[0] != UINT32_MAX)
			{
				const uint32_t Htarg = ptarget[7];

				uint32_t _ALIGN(64) vhash[8];
				be32enc(&endiandata[19], work->nonces[0]);
				x11khash(vhash, endiandata);
	
				if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
					work->valid_nonces = 1;
					work_set_target_ratio(work, vhash);
					work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
					if (work->nonces[1] != 0) {
						be32enc(&endiandata[19], work->nonces[1]);
						x11khash(vhash, endiandata);
						bn_set_target_ratio(work, vhash, 1);
						pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
						gpulog(LOG_DEBUG, thr_id, "second nonce %08x! cursor %08x", work->nonces[1], pdata[19]);
						work->valid_nonces++;
					} else {
						pdata[19] = work->nonces[0] + 1; // cursor
					}
					return work->valid_nonces;
				} else {
					gpu_increment_reject(thr_id);
					if (!opt_quiet)
						gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);

					pdata[19] = work->nonces[0] + 1;
					continue;
				}
			}
	
			if ((uint64_t) throughput + pdata[19] >= max_nonce) {
				pdata[19] = max_nonce;
				break;
			}
			pdata[19] += throughput;
	
		} while (!work_restart[thr_id].restart);

		*hashes_done = pdata[19] - first_nonce;
		return 0;
	}
	
	// cleanup
	extern "C" void free_x11k(int thr_id)
	{
		if (!init[thr_id])
			return;
	
		hipDeviceSynchronize();
	
		hipFree(d_hash[thr_id]);
		hipFree(seed_index);
	
		quark_blake512_cpu_free(thr_id);
		quark_groestl512_cpu_free(thr_id);
		x11_simd512_cpu_free(thr_id);
	
		cuda_check_cpu_free(thr_id);
		init[thr_id] = false;
	
		hipDeviceSynchronize();
	}